#include "hip/hip_runtime.h"
/*
 Copyright (c) 2025, The Neko Authors
 All rights reserved.

 Redistribution and use in source and binary forms, with or without
 modification, are permitted provided that the following conditions
 are met:

   * Redistributions of source code must retain the above copyright
     notice, this list of conditions and the following disclaimer.

   * Redistributions in binary form must reproduce the above
     copyright notice, this list of conditions and the following
     disclaimer in the documentation and/or other materials provided
     with the distribution.

   * Neither the name of the authors nor the names of its
     contributors may be used to endorse or promote products derived
     from this software without specific prior written permission.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 POSSIBILITY OF SUCH DAMAGE.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <device_config.h>
#include <device/cuda/check.h>
#include <math/bcknd/device/cuda/mathops_kernel.h>

/**
 * Device kernel for FST
 */
template< typename T, typename U>
__global__ void fst_kernel(
                           const U t,
                           const U Uinf,
                           T * __restrict__ u,
                           T * __restrict__ v,
                           T * __restrict__ w,
                           const int * __restrict__ mask,
                           const int n_mask,
                           const T * __restrict__ ubf,
                           const T * __restrict__ vbf,
                           const T * __restrict__ wbf,
                           const U * __restrict__ k_x,
                           const int n_total_modes,
                           const U * __restrict__ phi_0,
                           const int * __restrict__ shell,
                           const U * __restrict__ shell_amp,
                           const U * __restrict__ randvec,
                           const U cosa,
                           const U sina,
                           const U fringe_time,
                           const U * __restrict__ fs
                           ) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  U ra, rb, rc;
  U phi = 0.0;
  U phi_t = Uinf*t;
  U pert = 0.0;
  U f = 0.0;
  //int shellno;

  // phi_0 is size (k_length, n_mask) in fortran, so that means size
  // [n_mask][k_length] in C and therefore.

  for (int i = idx; i < n_mask; i += str) {

    ra = 0.0;
    rb = 0.0;
    rc = 0.0;
    for (int m = 0; m < n_total_modes; m += 1) {

      phi = phi_0[m + i*n_total_modes] - k_x[m] * phi_t;
      //shellno = shell[m];

      pert = shell_amp[shell[m]-1]*sin(phi);
      //ra += randvec[m + 0*n_total_modes]*pert;
      //rb += randvec[m + 1*n_total_modes]*pert;
      //rc += randvec[m + 2*n_total_modes]*pert;
      ra += randvec[m + 0*n_total_modes]*pert;
      rb += randvec[m + 1*n_total_modes]*pert;
      rc += randvec[m + 2*n_total_modes]*pert;
    }

    f = fringe_time*fs[i];
    u[mask[i+1]-1] = ubf[i] + f*ra;
    v[mask[i+1]-1] = vbf[i] + f*rb;
    w[mask[i+1]-1] = wbf[i] + f*rc;

  }
}

extern "C" {

void cuda_fst(real *t, real *Uinf,
              void *u_d, void *v_d, void *w_d, int *mask_d, int *n_mask,
              void *ubf_d, void *vbf_d, void *wbf_d, void *k_x_d,
              int *n_total_modes, void *phi_0_d, int *shell_d,
              void *shell_amp_d, void *randvec_d, real *cosa, real *sina,
              real *fringe_time, void *fs_d) {

  const dim3 nthrds(1024, 1, 1);
  const dim3 nblcks(*n_mask, 1, 1);
  const hipStream_t stream = (hipStream_t) glb_cmd_queue;

  fst_kernel<real, real>
    <<<nblcks, nthrds, 0, stream>>>(
                                    *t, *Uinf,
                                    (real *) u_d, (real *) v_d, (real *) w_d,
                                    (int *) mask_d, *n_mask,
                                    (real *) ubf_d, (real *) vbf_d, (real *) wbf_d,
                                    (real *) k_x_d, *n_total_modes,
                                    (real *) phi_0_d, (int *) shell_d, (real *) shell_amp_d,
                                    (real *) randvec_d,
                                    *cosa, *sina, *fringe_time, (real *) fs_d
                                    );
  CUDA_CHECK(hipGetLastError());

}

}
